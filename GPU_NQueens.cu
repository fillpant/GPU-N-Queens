#include "hip/hip_runtime.h"
#include "GPU_NQueens.cuh"
#include "deffinitions.cuh"
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>
#include <math.h>
#include "n_queens.cuh"
#include "assert.h"
#include "nq_utils.cuh"


__constant__ unsigned locked_row_end;

static bool check_gpu_compatibility(unsigned id, size_t size_of_states) {
	struct hipDeviceProp_t gpuprop;
	CHECK_CUDA_ERROR(hipGetDeviceProperties(&gpuprop, id));

	//alignment penalty is usually 256
	if (gpuprop.totalGlobalMem - 256 <= size_of_states) {
		fprintf(stderr, "Device %u (%s) does not have enough memory space (needed: %s).\n", id, gpuprop.name, util_size_to_human_readable(size_of_states));
		return false;
	}

	if (gpuprop.major * 100 + gpuprop.minor * 10 < MINIMUM_COMPUTE_CAPABILITY) {
		fprintf(stderr, "This program requires compute %.1f but device %u (%s) supports up to %u.%u.\n", MINIMUM_COMPUTE_CAPABILITY / 100.0, id, gpuprop.name, gpuprop.major, gpuprop.minor);
		return false;
	}

	if (gpuprop.warpSize != WARP_SIZE) {
		fprintf(stderr, "Device %u (%s) has a warp size of %u, however this program is compiled under the assumption that a warp contains %u threads.\n", id, gpuprop.name, gpuprop.warpSize, WARP_SIZE);
		return false;
	}

	if (gpuprop.computeMode == hipComputeModeProhibited) {
		fprintf(stderr, "Device %u (%s) cannot be accessed by this process.\n", id, gpuprop.name);
		return false;
	}

	if (gpuprop.sharedMemPerBlock < sizeof(nq_state_t) * COMPLETE_KERNEL_BLOCK_THREAD_COUNT) {
		fprintf(stderr, "Device %u (%s) doesn't have enough shared memory per block for %llu states.\n", id, gpuprop.name, (unsigned long long int)COMPLETE_KERNEL_BLOCK_THREAD_COUNT);
		return false;
	}

	return true;
}

__host__ uint64_t gpu_solver_driver(nq_state_t* const states, const uint_least32_t state_cnt, const unsigned row_locked, const gpu_config_t* const configs, const unsigned config_cnt) {
	FAIL_IF(!states);
	FAIL_IF(state_cnt == 0);
	FAIL_IF(!configs);
	FAIL_IF(config_cnt == 0);

	// Make sure all GPUs are capable of running this computation
	for (unsigned gpuc = 0; gpuc < config_cnt; ++gpuc) {
		//Code dup...
		const unsigned states_per_device = (unsigned)floor(state_cnt / config_cnt) + (gpuc == config_cnt - 1 ? state_cnt % config_cnt : 0);
		const unsigned padded_states_per_device = (states_per_device % 32 == 0 ? states_per_device : (states_per_device + (WARP_SIZE - states_per_device % WARP_SIZE)));
		FAIL_IF(!check_gpu_compatibility(configs[gpuc].device_id, sizeof(nq_state_t) * padded_states_per_device));
	}

	typedef struct { nq_state_t* d_states; unsigned d_statecnt, d_statecnt_padded, block_count; unsigned* d_results; } gpudata_t;

	gpudata_t* gdata = (gpudata_t*)calloc(config_cnt, sizeof(gpudata_t));

	nq_state_t* tmp_states = states;

	// Prepare and launch on each gpu. 
	for (unsigned gpuc = 0; gpuc < config_cnt; ++gpuc) {
		CHECK_CUDA_ERROR(hipSetDevice(configs[gpuc].device_id));

		//TODO check device capabilities match requirements (compute, memory, etc)!
		//TODO error handling leaves allocated memory on some devices.

		// Last device gets extra workload.
		const unsigned states_per_device = (unsigned)floor(state_cnt / config_cnt) + (gpuc == config_cnt - 1 ? state_cnt % config_cnt : 0);
		const unsigned padded_states_per_device = (states_per_device % 32 == 0 ? states_per_device : (states_per_device + (WARP_SIZE - states_per_device % WARP_SIZE)));
		const unsigned block_cnt_doublesweep_light_adv = (unsigned)ceil(states_per_device / (double)COMPLETE_KERNEL_BLOCK_THREAD_COUNT);

		printf("Preparing device %u...\n", configs[gpuc].device_id);

		nq_state_t* d_states = 0;
		unsigned* d_result = 0;
		CHECK_CUDA_ERROR(hipMalloc(&d_states, sizeof(nq_state_t) * padded_states_per_device));
		CHECK_CUDA_ERROR(hipMalloc(&d_result, sizeof(unsigned) * block_cnt_doublesweep_light_adv));
		if (configs[gpuc].async) {
			CHECK_CUDA_ERROR(hipMemsetAsync(d_states, 0, sizeof(nq_state_t) * padded_states_per_device));
			CHECK_CUDA_ERROR(hipMemcpyAsync(d_states, tmp_states, sizeof(nq_state_t) * states_per_device, hipMemcpyHostToDevice));
			CHECK_CUDA_ERROR(hipMemsetAsync(d_result, 0, sizeof(unsigned) * block_cnt_doublesweep_light_adv));
		} else {
			CHECK_CUDA_ERROR(hipMemset(d_states, 0, sizeof(nq_state_t) * padded_states_per_device));
			CHECK_CUDA_ERROR(hipMemcpy(d_states, tmp_states, sizeof(nq_state_t) * states_per_device, hipMemcpyHostToDevice));
			CHECK_CUDA_ERROR(hipMemset(d_result, 0, sizeof(unsigned) * block_cnt_doublesweep_light_adv));
		}
		tmp_states += states_per_device;

#ifndef __INTELLISENSE__ //Suppressing VS error...
		CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(locked_row_end), &row_locked, sizeof(unsigned)));
#endif
		gdata[gpuc].d_states = d_states;
		gdata[gpuc].d_statecnt = states_per_device;
		gdata[gpuc].d_statecnt_padded = padded_states_per_device;
		gdata[gpuc].d_results = d_result;
		gdata[gpuc].block_count = block_cnt_doublesweep_light_adv;
	}

	printf("Starting...\n");
	uint64_t result = 0;
	hipEvent_t ev = util_start_cuda_timer();
	unsigned max_blocks = 0;

#ifdef PROFILING_ROUNDS 
	float profiling_times[PROFILING_ROUNDS];
	for (unsigned profiling_run_counter = 0; profiling_run_counter < PROFILING_ROUNDS; ++profiling_run_counter) {
		max_blocks = 0;
		ev = util_start_cuda_timer();

#endif
		for (unsigned gpuc = 0; gpuc < config_cnt; ++gpuc) {
			CHECK_CUDA_ERROR(hipSetDevice(configs[gpuc].device_id));
			max_blocks = MAX(max_blocks, gdata[gpuc].block_count);
#ifdef USE_REGISTER_ONLY_KERNEL
			kern_doitall_v2_regld CUDA_KERNEL(gdata[gpuc].block_count, COMPLETE_KERNEL_BLOCK_THREAD_COUNT)(gdata[gpuc].d_states, gdata[gpuc].d_statecnt_padded, gdata[gpuc].d_results);
#else
			kern_doitall_v2_smem CUDA_KERNEL(gdata[gpuc].block_count, COMPLETE_KERNEL_BLOCK_THREAD_COUNT)(gdata[gpuc].d_states, gdata[gpuc].d_statecnt_padded, gdata[gpuc].d_results);
#endif

		}
		for (unsigned gpuc = 0; gpuc < config_cnt; ++gpuc) {
			CHECK_CUDA_ERROR(hipSetDevice(configs[gpuc].device_id));
			CHECK_CUDA_ERROR(hipDeviceSynchronize());
		}
		float time = util_end_event_get_time(ev);
		char* tmp_buf = util_milliseconds_to_duration((uint64_t)time);
		printf("\n\nComputation completed. Time taken: %.4fms. (%s)\n", time, tmp_buf);
		free(tmp_buf);
#ifdef PROFILING_ROUNDS
		profiling_times[profiling_run_counter] = time;
	}
	printf(">>> DATA:  ");
	float sum = 0;
	printf("%u %llu %u %u ", N, state_cnt, row_locked, COMPLETE_KERNEL_BLOCK_THREAD_COUNT);
	for (unsigned time = 0; time < PROFILING_ROUNDS; ++time) {
		printf("%.2f%c", profiling_times[time], time + 1 < PROFILING_ROUNDS ? ' ' : '\n');
		sum += profiling_times[time];
	}
	printf(">>> Total time: %.2fms Avg time: %.2fms\n", sum, sum / PROFILING_ROUNDS);
	// We don't clear the gpu buffers or anything so results are likely a multiple of the profiling rounds.
	printf(">>> No host-result summarisation. During profiling results may be inaccurate!\n");
	return result;
#else
		unsigned* per_block_results;
		CHECK_CUDA_ERROR(hipHostMalloc(&per_block_results, sizeof(unsigned) * max_blocks));
		for (unsigned gpuc = 0; gpuc < config_cnt; ++gpuc) {
			CHECK_CUDA_ERROR(hipSetDevice(configs[gpuc].device_id));
			CHECK_CUDA_ERROR(hipMemcpy(per_block_results, gdata[gpuc].d_results, sizeof(unsigned) * gdata[gpuc].block_count, hipMemcpyDeviceToHost));
			for (unsigned a = 0; a < gdata[gpuc].block_count; ++a)
				result += per_block_results[a];
			CHECK_CUDA_ERROR(hipFree(gdata[gpuc].d_states));
			CHECK_CUDA_ERROR(hipFree(gdata[gpuc].d_results));
		}
		CHECK_CUDA_ERROR(hipHostFree(per_block_results));
		return result;
#endif
}
#ifdef USE_REGISTER_ONLY_KERNEL

	__global__ void kern_doitall_v2_regld_old(const nq_state_t* const __restrict__ states, const unsigned state_cnt, unsigned* const __restrict__ sols) {
		const unsigned local_idx = threadIdx.x;
		const unsigned global_idx = blockIdx.x * blockDim.x + local_idx;
		__shared__ unsigned char smem[COMPLETE_KERNEL_BLOCK_THREAD_COUNT * N + sizeof(unsigned int) * WARP_SIZE];
		register unsigned t_sols = 0;

		if (global_idx < state_cnt) {
			unsigned char* const __restrict__ l_smem = smem + local_idx * N;
			// Since we have relatively low register pressure (on tested architectures) we can make use of the spare registers as 'memory space' for each thread 
			// instead of shared memory. Struct is broken down to components (hopefully) placed in registers as below:
			register bitset32_t queens_in_columns = states[global_idx].queens_in_columns;
			register uint64_t diagonal = states[global_idx].diagonals.diagonal, antidiagonal = states[global_idx].diagonals.antidiagonal;
			register int curr_row = states[global_idx].curr_row;
			//The queens at index array cannot be placed in a register (without a lot of effort and preprocessor 'hacks' that is) so it stays in smem.
#pragma unroll
			for (int i = 0; i < N; ++i)
				l_smem[i] = states[global_idx].queen_at_index[i];

			do {
				int res = curr_row >= locked_row_end;
				if (!__ballot_sync(0xFFFFFFFF, res))
					break; // Whole warp finished
				if (res) {
					//NOTE: In an effort to speed 
					// using 'find nth bit' (FNB) results in significantly poorer performance than conditionally shifting
					//Advance the state
					while (curr_row >= locked_row_end) {
						const register unsigned queen_index = l_smem[curr_row];
						bitset32_t free_cols = (~(queens_in_columns | dad_extract_explicit(diagonal, antidiagonal, curr_row)) & N_MASK);
						if (queen_index != UNSET_QUEEN_INDEX) {
							// Tried to change the logic to issue a single FNB (Find Nth Bit) instruction, depending on the position of the queen 
							free_cols &= (N_MASK << (queen_index + 1));
							queens_in_columns = bs_clear_bit(queens_in_columns, queen_index);
							l_smem[curr_row] = UNSET_QUEEN_INDEX;
							diagonal &= ~((1LLU << queen_index) << curr_row);
							antidiagonal &= ~((1LLU << queen_index) << (64 - N - curr_row));
						}
						if (!free_cols) {
							--curr_row;
						} else {
							//direct ffs is okay here, free_cols will have at least one set bit.
							const unsigned col = intrin_ffs_nosub(free_cols);
							queens_in_columns = bs_set_bit(queens_in_columns, col);
							l_smem[curr_row] = col;
							diagonal |= ((uint64_t)1U << col) << curr_row;
							antidiagonal |= ((uint64_t)1U << col) << (64 - N - curr_row);
							if (curr_row < N - 1)
								++curr_row;
							break;
						}
					}
				}

				__syncwarp();

				if (res) {
					while (l_smem[curr_row] == UNSET_QUEEN_INDEX) {
						const bitset32_t free_cols = (~(queens_in_columns | dad_extract_explicit(diagonal, antidiagonal, curr_row)) & N_MASK);
						const int POPCNT(free_cols, popcnt);
						if (popcnt == 1) {
#ifdef NQ_ENABLE_EXPERIMENTAL_OPTIMISATIONS
							const unsigned col = intrin_find_leading_one_u32(free_cols);
#else
							const unsigned col = __ffs(free_cols) + 1;
#endif
							queens_in_columns = bs_set_bit(queens_in_columns, col);
							l_smem[curr_row] = col;
							diagonal |= ((uint64_t)1U << col) << curr_row;
							antidiagonal |= ((uint64_t)1U << col) << (64 - N - curr_row);
							if (curr_row < N - 1) ++curr_row;
						} else break;
					}
				}
				__syncwarp();
				t_sols += (queens_in_columns == N_MASK);
			} while (1);
		}
		__syncthreads();
		t_sols = block_reduce_sum_shfl_variwarp((unsigned)t_sols, (unsigned int*)&smem[COMPLETE_KERNEL_BLOCK_THREAD_COUNT * N]);

		if (!local_idx)
			sols[blockIdx.x] += t_sols;
	}




	__global__ void kern_doitall_v2_regld(const nq_state_t* const __restrict__ states, const unsigned state_cnt, unsigned* const __restrict__ sols) {
		const unsigned local_idx = threadIdx.x;
		const unsigned global_idx = blockIdx.x * blockDim.x + local_idx;
		__shared__ unsigned char smem[COMPLETE_KERNEL_BLOCK_THREAD_COUNT * N + sizeof(unsigned int) * WARP_SIZE];
		register unsigned t_sols = 0;

		if (global_idx < state_cnt) {
			register unsigned int l_reg_0 = states[global_idx].queen_at_index[0]

#if N>=1
				, l_reg_1 = states[global_idx].queen_at_index[1]
#endif
#if N>=2
				, l_reg_2 = states[global_idx].queen_at_index[2]
#endif 
#if N>=3
				, l_reg_3 = states[global_idx].queen_at_index[3]
#endif
#if N>=4
				, l_reg_4 = states[global_idx].queen_at_index[4]
#endif
#if N>=5
				, l_reg_5 = states[global_idx].queen_at_index[5]
#endif
#if N>=6
				, l_reg_6 = states[global_idx].queen_at_index[6]
#endif
#if N>=7
				, l_reg_7 = states[global_idx].queen_at_index[7]
#endif
#if N>=8
				, l_reg_8 = states[global_idx].queen_at_index[8]
#endif
#if N>=9
				, l_reg_9 = states[global_idx].queen_at_index[9]
#endif
#if N>=10
				, l_reg_10 = states[global_idx].queen_at_index[10]
#endif
#if N>=11
				, l_reg_11 = states[global_idx].queen_at_index[11]
#endif
#if N>=12
				, l_reg_12 = states[global_idx].queen_at_index[12]
#endif
#if N>=13
				, l_reg_13 = states[global_idx].queen_at_index[13]
#endif
#if N>=14
				, l_reg_14 = states[global_idx].queen_at_index[14]
#endif
#if N>=15
				, l_reg_15 = states[global_idx].queen_at_index[15]
#endif
#if N>=16
				, l_reg_16 = states[global_idx].queen_at_index[16]
#endif
#if N>=17
				, l_reg_17 = states[global_idx].queen_at_index[17]
#endif
#if N>=18
				, l_reg_18 = states[global_idx].queen_at_index[18]
#endif
#if N>=19
				, l_reg_19 = states[global_idx].queen_at_index[19]
#endif
#if N>=20
				, l_reg_20 = states[global_idx].queen_at_index[20]
#endif
#if N>=21
				, l_reg_21 = states[global_idx].queen_at_index[21]
#endif
#if N>=22
				, l_reg_22 = states[global_idx].queen_at_index[22]
#endif
#if N>=23
				, l_reg_23 = states[global_idx].queen_at_index[23]
#endif
#if N>=24
				, l_reg_24 = states[global_idx].queen_at_index[24]
#endif
#if N>=25
				, l_reg_25 = states[global_idx].queen_at_index[25]
#endif
#if N>=26
				, l_reg_26 = states[global_idx].queen_at_index[26]
#endif
#if N>=27
				, l_reg_27 = states[global_idx].queen_at_index[27]
#endif
#if N>=28
				, l_reg_28 = states[global_idx].queen_at_index[28]
#endif
#if N>=29
				, l_reg_29 = states[global_idx].queen_at_index[29]
#endif
#if N>=30
				, l_reg_30 = states[global_idx].queen_at_index[30]
#endif
#if N>=31
				, l_reg_31 = states[global_idx].queen_at_index[31]
#endif

#if N>=32
				, l_reg_32 = states[global_idx].queen_at_index[32]
#endif				
#if N>=33
#error "More ifs needed here..."
#endif
				;
			unsigned char* const __restrict__ l_smem = smem + local_idx * N;
			// Since we have relatively low register pressure (on tested architectures) we can make use of the spare registers as 'memory space' for each thread 
			// instead of shared memory. Struct is broken down to components (hopefully) placed in registers as below:
			register bitset32_t queens_in_columns = states[global_idx].queens_in_columns;
			register uint64_t diagonal = states[global_idx].diagonals.diagonal, antidiagonal = states[global_idx].diagonals.antidiagonal;
			register int curr_row = states[global_idx].curr_row;
			
//			#pragma unroll
//			for (int i = 0; i < N; ++i)
//				l_smem[i] = states[global_idx].queen_at_index[i];

			do {
				int res = curr_row >= locked_row_end;
				if (!__ballot_sync(0xFFFFFFFF, res))
					break; // Whole warp finished
			
				if (res) {
					while (curr_row >= locked_row_end) {
						register unsigned queen_index;

#if N >= 1
						if (curr_row == 0) queen_index = l_reg_0;
#endif
#if N >= 2
						if (curr_row == 1) queen_index = l_reg_1;
#endif
#if N >= 3
						if (curr_row == 2) queen_index = l_reg_2;
#endif
#if N >= 4
						if (curr_row == 3) queen_index = l_reg_3;
#endif
#if N >= 5
						if (curr_row == 4) queen_index = l_reg_4;
#endif
#if N >= 6
						if (curr_row == 5) queen_index = l_reg_5;
#endif
#if N >= 7
						if (curr_row == 6) queen_index = l_reg_6;
#endif
#if N >= 8
						if (curr_row == 7) queen_index = l_reg_7;
#endif
#if N >= 9
						if (curr_row == 8) queen_index = l_reg_8;
#endif
#if N >= 10
						if (curr_row == 9) queen_index = l_reg_9;
#endif
#if N >= 11
						if (curr_row == 10) queen_index = l_reg_10;
#endif
#if N >= 12
						if (curr_row == 11) queen_index = l_reg_11;
#endif
#if N >= 13
						if (curr_row == 12) queen_index = l_reg_12;
#endif
#if N >= 14
						if (curr_row == 13) queen_index = l_reg_13;
#endif
#if N >= 15
						if (curr_row == 14) queen_index = l_reg_14;
#endif
#if N >= 16
						if (curr_row == 15) queen_index = l_reg_15;
#endif
#if N >= 17
						if (curr_row == 16) queen_index = l_reg_16;
#endif
#if N >= 18
						if (curr_row == 17) queen_index = l_reg_17;
#endif
#if N >= 19
						if (curr_row == 18) queen_index = l_reg_18;
#endif
#if N >= 20
						if (curr_row == 19) queen_index = l_reg_19;
#endif
#if N >= 21
						if (curr_row == 20) queen_index = l_reg_20;
#endif
#if N >= 22
						if (curr_row == 21) queen_index = l_reg_21;
#endif
#if N >= 23
						if (curr_row == 22) queen_index = l_reg_22;
#endif
#if N >= 24
						if (curr_row == 23) queen_index = l_reg_23;
#endif
#if N >= 25
						if (curr_row == 24) queen_index = l_reg_24;
#endif
#if N >= 26
						if (curr_row == 25) queen_index = l_reg_25;
#endif
#if N >= 27
						if (curr_row == 26) queen_index = l_reg_26;
#endif
#if N >= 28
						if (curr_row == 27) queen_index = l_reg_27;
#endif
#if N >= 29
						if (curr_row == 28) queen_index = l_reg_28;
#endif
#if N >= 30
						if (curr_row == 29) queen_index = l_reg_29;
#endif
#if N >= 31
						if (curr_row == 30) queen_index = l_reg_30;
#endif
#if N >= 32
						if (curr_row == 31) queen_index = l_reg_31;
#endif				
#if N >= 33
#error "More ifs needed here too..."
#endif
						bitset32_t free_cols = (~(queens_in_columns | dad_extract_explicit(diagonal, antidiagonal, curr_row)) & N_MASK);
						if (queen_index != UNSET_QUEEN_INDEX) {
							// Tried to change the logic to issue a single FNB (Find Nth Bit) instruction, depending on the position of the queen 
							free_cols &= (N_MASK << (queen_index + 1));
							queens_in_columns = bs_clear_bit(queens_in_columns, queen_index);

#if N >= 1
							if (curr_row == 0) l_reg_0 = UNSET_QUEEN_INDEX;
#endif
#if N >= 2
							else if (curr_row == 1) l_reg_1 = UNSET_QUEEN_INDEX;
#endif
#if N >= 3
							else if (curr_row == 2) l_reg_2 = UNSET_QUEEN_INDEX;
#endif
#if N >= 4
							else if (curr_row == 3) l_reg_3 = UNSET_QUEEN_INDEX;
#endif
#if N >= 5
							else if (curr_row == 4) l_reg_4 = UNSET_QUEEN_INDEX;
#endif
#if N >= 6
							else if (curr_row == 5) l_reg_5 = UNSET_QUEEN_INDEX;
#endif
#if N >= 7
							else if (curr_row == 6) l_reg_6 = UNSET_QUEEN_INDEX;
#endif
#if N >= 8
							else if (curr_row == 7) l_reg_7 = UNSET_QUEEN_INDEX;
#endif
#if N >= 9
							else if (curr_row == 8) l_reg_8 = UNSET_QUEEN_INDEX;
#endif
#if N >= 10
							else if (curr_row == 9) l_reg_9 = UNSET_QUEEN_INDEX;
#endif
#if N >= 11
							else if (curr_row == 10) l_reg_10 = UNSET_QUEEN_INDEX;
#endif
#if N >= 12
							else if (curr_row == 11) l_reg_11 = UNSET_QUEEN_INDEX;
#endif
#if N >= 13
							else if (curr_row == 12) l_reg_12 = UNSET_QUEEN_INDEX;
#endif
#if N >= 14
							else if (curr_row == 13) l_reg_13 = UNSET_QUEEN_INDEX;
#endif
#if N >= 15
							else if (curr_row == 14) l_reg_14 = UNSET_QUEEN_INDEX;
#endif
#if N >= 16
							else if (curr_row == 15) l_reg_15 = UNSET_QUEEN_INDEX;
#endif
#if N >= 17
							else if (curr_row == 16) l_reg_16 = UNSET_QUEEN_INDEX;
#endif
#if N >= 18
							else if (curr_row == 17) l_reg_17 = UNSET_QUEEN_INDEX;
#endif
#if N >= 19
							else if (curr_row == 18) l_reg_18 = UNSET_QUEEN_INDEX;
#endif
#if N >= 20
							else if (curr_row == 19) l_reg_19 = UNSET_QUEEN_INDEX;
#endif
#if N >= 21
							else if (curr_row == 20) l_reg_20 = UNSET_QUEEN_INDEX;
#endif
#if N >= 22
							else if (curr_row == 21) l_reg_21 = UNSET_QUEEN_INDEX;
#endif
#if N >= 23
							else if (curr_row == 22) l_reg_22 = UNSET_QUEEN_INDEX;
#endif
#if N >= 24
							else if (curr_row == 23) l_reg_23 = UNSET_QUEEN_INDEX;
#endif
#if N >= 25
							else if (curr_row == 24) l_reg_24 = UNSET_QUEEN_INDEX;
#endif
#if N >= 26
							else if (curr_row == 25) l_reg_25 = UNSET_QUEEN_INDEX;
#endif
#if N >= 27
							else if (curr_row == 26) l_reg_26 = UNSET_QUEEN_INDEX;
#endif
#if N >= 28
							else if (curr_row == 27) l_reg_27 = UNSET_QUEEN_INDEX;
#endif
#if N >= 29
							else if (curr_row == 28) l_reg_28 = UNSET_QUEEN_INDEX;
#endif
#if N >= 30
							else if (curr_row == 29) l_reg_29 = UNSET_QUEEN_INDEX;
#endif
#if N >= 31
							else if (curr_row == 30) l_reg_30 = UNSET_QUEEN_INDEX;
#endif
#if N >= 32
							else if (curr_row == 31) l_reg_31 = UNSET_QUEEN_INDEX;
#endif				
#if N >= 33
#error "More ifs needed here too..."
#endif
							diagonal &= ~((1LLU << queen_index) << curr_row);
							antidiagonal &= ~((1LLU << queen_index) << (64 - N - curr_row));
						}
						if (!free_cols) {
							--curr_row;
						} else {
							//direct ffs is okay here, free_cols will have at least one set bit.
							const unsigned col = intrin_ffs_nosub(free_cols);
							queens_in_columns = bs_set_bit(queens_in_columns, col);
#if N >= 1
							if (curr_row == 0) l_reg_0 = col;
#endif
#if N >= 2
							else if (curr_row == 1) l_reg_1 = col;
#endif
#if N >= 3
							else if (curr_row == 2) l_reg_2 = col;
#endif
#if N >= 4
							else if (curr_row == 3) l_reg_3 = col;
#endif
#if N >= 5
							else if (curr_row == 4) l_reg_4 = col;
#endif
#if N >= 6
							else if (curr_row == 5) l_reg_5 = col;
#endif
#if N >= 7
							else if (curr_row == 6) l_reg_6 = col;
#endif
#if N >= 8
							else if (curr_row == 7) l_reg_7 = col;
#endif
#if N >= 9
							else if (curr_row == 8) l_reg_8 = col;
#endif
#if N >= 10
							else if (curr_row == 9) l_reg_9 = col;
#endif
#if N >= 11
							else if (curr_row == 10) l_reg_10 = col;
#endif
#if N >= 12
							else if (curr_row == 11) l_reg_11 = col;
#endif
#if N >= 13
							else if (curr_row == 12) l_reg_12 = col;
#endif
#if N >= 14
							else if (curr_row == 13) l_reg_13 = col;
#endif
#if N >= 15
							else if (curr_row == 14) l_reg_14 = col;
#endif
#if N >= 16
							else if (curr_row == 15) l_reg_15 = col;
#endif
#if N >= 17
							else if (curr_row == 16) l_reg_16 = col;
#endif
#if N >= 18
							else if (curr_row == 17) l_reg_17 = col;
#endif
#if N >= 19
							else if (curr_row == 18) l_reg_18 = col;
#endif
#if N >= 20
							else if (curr_row == 19) l_reg_19 = col;
#endif
#if N >= 21
							else if (curr_row == 20) l_reg_20 = col;
#endif
#if N >= 22
							else if (curr_row == 21) l_reg_21 = col;
#endif
#if N >= 23
							else if (curr_row == 22) l_reg_22 = col;
#endif
#if N >= 24
							else if (curr_row == 23) l_reg_23 = col;
#endif
#if N >= 25
							else if (curr_row == 24) l_reg_24 = col;
#endif
#if N >= 26
							else if (curr_row == 25) l_reg_25 = col;
#endif
#if N >= 27
							else if (curr_row == 26) l_reg_26 = col;
#endif
#if N >= 28
							else if (curr_row == 27) l_reg_27 = col;
#endif
#if N >= 29
							else if (curr_row == 28) l_reg_28 = col;
#endif
#if N >= 30
							else if (curr_row == 29) l_reg_29 = col;
#endif
#if N >= 31
							else if (curr_row == 30) l_reg_30 = col;
#endif
#if N >= 32
							else if (curr_row == 31) l_reg_31 = col;
#endif				
#if N >= 33
#error "More ifs needed here too..."
#endif
							diagonal |= ((uint64_t)1U << col) << curr_row;
							antidiagonal |= ((uint64_t)1U << col) << (64 - N - curr_row);
							if (curr_row < N - 1)
								++curr_row;
							break;
						}
					}
				}

				__syncwarp();

				if (res) {
					while (
#if N >= 1
						curr_row == 0 ? l_reg_0 == UNSET_QUEEN_INDEX : 
#endif
#if N >= 2
						curr_row == 1 ? l_reg_1 == UNSET_QUEEN_INDEX :
#endif
#if N >= 3
						curr_row == 2 ? l_reg_2 == UNSET_QUEEN_INDEX :
#endif
#if N >= 4
						curr_row == 3 ? l_reg_3 == UNSET_QUEEN_INDEX :
#endif
#if N >= 5
						curr_row == 4 ? l_reg_4 == UNSET_QUEEN_INDEX :
#endif
#if N >= 6
						curr_row == 5 ? l_reg_5 == UNSET_QUEEN_INDEX :
#endif
#if N >= 7
						curr_row == 6 ? l_reg_6 == UNSET_QUEEN_INDEX :
#endif
#if N >= 8
						curr_row == 7 ? l_reg_7 == UNSET_QUEEN_INDEX :
#endif
#if N >= 9
						curr_row == 8 ? l_reg_8 == UNSET_QUEEN_INDEX :
#endif
#if N >= 10
						curr_row == 9 ? l_reg_9 == UNSET_QUEEN_INDEX :
#endif
#if N >= 11
						curr_row == 10 ? l_reg_10 == UNSET_QUEEN_INDEX :
#endif
#if N >= 12
						curr_row == 11 ? l_reg_11 == UNSET_QUEEN_INDEX :
#endif
#if N >= 13
						curr_row == 12 ? l_reg_12 == UNSET_QUEEN_INDEX :
#endif
#if N >= 14
						curr_row == 13 ? l_reg_13 == UNSET_QUEEN_INDEX :
#endif
#if N >= 15
						curr_row == 14 ? l_reg_14 == UNSET_QUEEN_INDEX :
#endif
#if N >= 16
						curr_row == 15 ? l_reg_15 == UNSET_QUEEN_INDEX :
#endif
#if N >= 17
						curr_row == 16 ? l_reg_16 == UNSET_QUEEN_INDEX :
#endif
#if N >= 18
						curr_row == 17 ? l_reg_17 == UNSET_QUEEN_INDEX :
#endif
#if N >= 19
						curr_row == 18 ? l_reg_18 == UNSET_QUEEN_INDEX :
#endif
#if N >= 20
						curr_row == 19 ? l_reg_19 == UNSET_QUEEN_INDEX :
#endif
#if N >= 21
						curr_row == 20 ? l_reg_20 == UNSET_QUEEN_INDEX :
#endif
#if N >= 22
						curr_row == 21 ? l_reg_21 == UNSET_QUEEN_INDEX :
#endif
#if N >= 23
						curr_row == 22 ? l_reg_22 == UNSET_QUEEN_INDEX :
#endif
#if N >= 24
						curr_row == 23 ? l_reg_23 == UNSET_QUEEN_INDEX :
#endif
#if N >= 25
						curr_row == 24 ? l_reg_24 == UNSET_QUEEN_INDEX :
#endif
#if N >= 26
						curr_row == 25 ? l_reg_25 == UNSET_QUEEN_INDEX :
#endif
#if N >= 27
						curr_row == 26 ? l_reg_26 == UNSET_QUEEN_INDEX :
#endif
#if N >= 28
						curr_row == 27 ? l_reg_27 == UNSET_QUEEN_INDEX :
#endif
#if N >= 29
						curr_row == 28 ? l_reg_28 == UNSET_QUEEN_INDEX :
#endif
#if N >= 30
						curr_row == 29 ? l_reg_29 == UNSET_QUEEN_INDEX :
#endif
#if N >= 31
						curr_row == 30 ? l_reg_30 == UNSET_QUEEN_INDEX :
#endif
#if N >= 32
						curr_row == 31 ? l_reg_31 == UNSET_QUEEN_INDEX :
#endif				
#if N >= 33
#error "More ifs needed here too..."
#endif
						0
						
						) {
						const bitset32_t free_cols = (~(queens_in_columns | dad_extract_explicit(diagonal, antidiagonal, curr_row)) & N_MASK);
						const int POPCNT(free_cols, popcnt);
						if (popcnt == 1) {
#ifdef NQ_ENABLE_EXPERIMENTAL_OPTIMISATIONS
							const unsigned col = intrin_find_leading_one_u32(free_cols);
#else
							const unsigned col = __ffs(free_cols) + 1;
#endif
							queens_in_columns = bs_set_bit(queens_in_columns, col);
#if N >= 1
							if (curr_row == 0) l_reg_0 = col;
#endif
#if N >= 2
							else if (curr_row == 1) l_reg_1 = col;
#endif
#if N >= 3
							else if (curr_row == 2) l_reg_2 = col;
#endif
#if N >= 4
							else if (curr_row == 3) l_reg_3 = col;
#endif
#if N >= 5
							else if (curr_row == 4) l_reg_4 = col;
#endif
#if N >= 6
							else if (curr_row == 5) l_reg_5 = col;
#endif
#if N >= 7
							else if (curr_row == 6) l_reg_6 = col;
#endif
#if N >= 8
							else if (curr_row == 7) l_reg_7 = col;
#endif
#if N >= 9
							else if (curr_row == 8) l_reg_8 = col;
#endif
#if N >= 10
							else if (curr_row == 9) l_reg_9 = col;
#endif
#if N >= 11
							else if (curr_row == 10) l_reg_10 = col;
#endif
#if N >= 12
							else if (curr_row == 11) l_reg_11 = col;
#endif
#if N >= 13
							else if (curr_row == 12) l_reg_12 = col;
#endif
#if N >= 14
							else if (curr_row == 13) l_reg_13 = col;
#endif
#if N >= 15
							else if (curr_row == 14) l_reg_14 = col;
#endif
#if N >= 16
							else if (curr_row == 15) l_reg_15 = col;
#endif
#if N >= 17
							else if (curr_row == 16) l_reg_16 = col;
#endif
#if N >= 18
							else if (curr_row == 17) l_reg_17 = col;
#endif
#if N >= 19
							else if (curr_row == 18) l_reg_18 = col;
#endif
#if N >= 20
							else if (curr_row == 19) l_reg_19 = col;
#endif
#if N >= 21
							else if (curr_row == 20) l_reg_20 = col;
#endif
#if N >= 22
							else if (curr_row == 21) l_reg_21 = col;
#endif
#if N >= 23
							else if (curr_row == 22) l_reg_22 = col;
#endif
#if N >= 24
							else if (curr_row == 23) l_reg_23 = col;
#endif
#if N >= 25
							else if (curr_row == 24) l_reg_24 = col;
#endif
#if N >= 26
							else if (curr_row == 25) l_reg_25 = col;
#endif
#if N >= 27
							else if (curr_row == 26) l_reg_26 = col;
#endif
#if N >= 28
							else if (curr_row == 27) l_reg_27 = col;
#endif
#if N >= 29
							else if (curr_row == 28) l_reg_28 = col;
#endif
#if N >= 30
							else if (curr_row == 29) l_reg_29 = col;
#endif
#if N >= 31
							else if (curr_row == 30) l_reg_30 = col;
#endif
#if N >= 32
							else if (curr_row == 31) l_reg_31 = col;
#endif				
#if N >= 33
#error "More ifs needed here too..."
#endif
							diagonal |= ((uint64_t)1U << col) << curr_row;
							antidiagonal |= ((uint64_t)1U << col) << (64 - N - curr_row);
							if (curr_row < N - 1) ++curr_row;
						} else break;
					}
				}
				__syncwarp();
				t_sols += (queens_in_columns == N_MASK);
			} while (1);
		}
		__syncthreads();
		t_sols = block_reduce_sum_shfl_variwarp((unsigned)t_sols, (unsigned int*)&smem[COMPLETE_KERNEL_BLOCK_THREAD_COUNT * N]);

		if (!local_idx)
			sols[blockIdx.x] += t_sols;
	}




#else 
	// Warning: state_cnt MUST be a multiple of 32 and states must be padded respectively.
	__global__ void kern_doitall_v2_smem(const nq_state_t* const __restrict__ states, const uint_least32_t state_cnt, unsigned* const __restrict__ sols) {
		const uint_least32_t local_idx = threadIdx.x;
		const uint_least32_t global_idx = blockIdx.x * blockDim.x + local_idx;
		__shared__ nq_state_t smem[COMPLETE_KERNEL_BLOCK_THREAD_COUNT + CEILING((sizeof(unsigned int) * WARP_SIZE), sizeof(nq_state_t))];
		register unsigned t_sols = 0;

		if (global_idx < state_cnt) {
			smem[local_idx].queens_in_columns = states[global_idx].queens_in_columns;
			smem[local_idx].diagonals = states[global_idx].diagonals;
			smem[local_idx].curr_row = states[global_idx].curr_row;
#pragma unroll
			for (int i = 0; i < N; ++i) {
				smem[local_idx].queen_at_index[i] = states[global_idx].queen_at_index[i];
			}
			__syncthreads();
			do {
				int res = smem[local_idx].curr_row >= locked_row_end;
				bool any_alive = __ballot_sync(0xFFFFFFFF, res);
				if (!any_alive) // Whole warp finished
					break;
				if (res) device_advance_nq_state(&smem[local_idx], locked_row_end);
				__syncwarp(); // Threads made to converge before doublesweep_light
				if (res)	device_doublesweep_light_nq_state(&smem[local_idx]);
				//__syncwarp(); // Threads made to converge before the following line
				t_sols += (smem[local_idx].queens_in_columns == N_MASK);//Non divergent
			//}
			//__syncwarp();
			} while (1);
		}
		__syncthreads();
		t_sols = block_reduce_sum_shfl_variwarp((unsigned)t_sols, (unsigned int*)&smem[COMPLETE_KERNEL_BLOCK_THREAD_COUNT]);

		if (!local_idx) {
			sols[blockIdx.x] += t_sols;
		}
	}
#endif

	__host__ nq_state_t* copy_states_to_gpu(const nq_state_t* const states, const uint64_t state_count, const gpu_config_t* const config) {
		CHECK_CUDA_ERROR(hipSetDevice(config->device_id));
		nq_state_t* d_states;
		CHECK_CUDA_ERROR(hipMalloc(&d_states, sizeof(nq_state_t) * state_count));
		if (config->async) {
			CHECK_CUDA_ERROR(hipMemcpy(d_states, states, sizeof(nq_state_t) * state_count, hipMemcpyHostToDevice));
		} else {
			CHECK_CUDA_ERROR(hipMemcpyAsync(d_states, states, sizeof(nq_state_t) * state_count, hipMemcpyHostToDevice));
		}
		return d_states;
	}

	__host__ void copy_states_from_gpu(nq_state_t* host_states, nq_state_t* device_states, const uint64_t state_count, const gpu_config_t* const config) {
		if (config->async) {
			CHECK_CUDA_ERROR(hipMemcpyAsync(host_states, device_states, state_count * sizeof(nq_state_t), hipMemcpyDeviceToHost));
		} else {
			CHECK_CUDA_ERROR(hipMemcpy(host_states, device_states, state_count * sizeof(nq_state_t), hipMemcpyDeviceToHost));
		}
	}
